#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define Threads 1000

#define readBlockSize 10000000

/*
	A little hackish once digits of pi exceed 67~ million. See lines 80-84.
	For digits of pi lower than that, commenting out lines 81-84 should do. Leave
	line 80 uncommented so printf and fprintf from lines 135-136 will remain functional.
*/

__global__ void computeHistogram(char*, int*);
__global__ void blankCall() {int i = 0; if (i == 0) {} };

int main(int argc, char *argv[]) {

	if (argc != 2) {
		printf("\nError: Number of arguments incorrect.\n"
			"There can only be 1 additional argument, which is the name of the file.\n"
			"Ex: ./Multi-Pi pi.txt\n"
			"Program gracefully terminated.\n");
		exit(0);
	}

	FILE *input = fopen(argv[1], "r+");
	FILE *output = fopen("freq.dat", "w+");
	if (input == NULL) {
		printf("Error: %s could not be read!\n"
			"Program gracefully terminated.\n", argv[1]);
		exit(1);
	}
	if (output == NULL) {
		printf("Error: freq.dat could not be created!\n"
				"Program gracefully terminated.\n");
		exit(1);
	}

	//------------------------------------------------------------------------------------//

	hipSetDevice(0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipSetDevice(1);
	hipStream_t stream2;
	hipStreamCreate(&stream2); 

	clock_t start, end;
	double exec_time;
	start = clock();

	//===================================================

	char* inputString1; char* inputString2; 
	int* histogram;
	int histogram2[10] = {0}, count = 0, count2 = 1, firstDigit; 
	char *dev_inputString1;
	char *dev_inputString2; 
	int *dev_histogram1;
	int *dev_histogram2;

	hipHostAlloc((void**) &inputString1, sizeof(char) * readBlockSize, hipHostMallocDefault);
	hipHostAlloc((void**) &inputString2, sizeof(char) * readBlockSize, hipHostMallocDefault);
	hipHostAlloc((void**) &histogram, sizeof(int) * 10, hipHostMallocDefault);
	for (int i = 0; i < 10; i++) {
		histogram[i] = 0;
		histogram2[i] = 0;
	}

	hipSetDevice(0);
		blankCall<<<1, 1>>>();
		hipMalloc((void**)&dev_inputString1, sizeof(char) * readBlockSize);
		hipMalloc((void**)&dev_histogram1, sizeof(int) * 10);
		hipMemcpy(dev_histogram1, histogram, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipSetDevice(1);
		blankCall<<<1, 1>>>();
		hipMalloc((void**)&dev_inputString2, sizeof(char) * readBlockSize);
		hipMalloc((void**)&dev_histogram2, sizeof(int) * 10);
		hipMemcpy(dev_histogram2, histogram, 10 * sizeof(int), hipMemcpyHostToDevice);

	int tmp[12] = {0};
	fgets(inputString1, 12, input); //12
	firstDigit = inputString1[0] - '0';
	for (int i = 0; i < 11; i++) {
		tmp[inputString1[i] - '0']++;
		printf("%i ", inputString1[i] - '0');
	}

	// for (int i = 0; i < 5; i++) {
	// 	printf("Run %i\n", i+1);
	// 	fgets(inputString1, readBlockSize, input);
	// 	hipSetDevice(0);
	// 	hipMemcpyAsync(dev_inputString1, inputString1, readBlockSize * sizeof(char), hipMemcpyHostToDevice, stream1);
	// 	computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, 0, stream1>>>(dev_inputString1, dev_histogram1);
		
	// 	fgets(inputString2, readBlockSize, input);
	// 	hipSetDevice(1);
	// 	hipMemcpyAsync(dev_inputString2, inputString2, readBlockSize * sizeof(char), hipMemcpyHostToDevice, stream2);
	// 	computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, 0, stream2>>>(dev_inputString2, dev_histogram2);
	// }


	while(fgets(inputString1, readBlockSize, input) != NULL) {
		hipSetDevice(0);
		// hipMemcpy(dev_inputString1, inputString1, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
		// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString1, dev_histogram1);
		hipMemcpyAsync(dev_inputString1, inputString1, readBlockSize * sizeof(char), hipMemcpyHostToDevice, stream1);
		computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, 0, stream1>>>(dev_inputString1, dev_histogram1);
		printf("GPU-0 (%i)\n", count); count = count + readBlockSize;

		if (fgets(inputString2, readBlockSize, input) != NULL) {
			hipSetDevice(1);
			// hipMemcpy(dev_inputString2, inputString2, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
			// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString2, dev_histogram2);
			hipMemcpyAsync(dev_inputString2, inputString2, readBlockSize * sizeof(char), hipMemcpyHostToDevice, stream2);
			computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, 0, stream2>>>(dev_inputString2, dev_histogram2);
			printf("GPU-1 (%i)\n", count); count = count + readBlockSize;
		}
		// printf("GPUs Synchronized (%i)\n", count);
		// count++;
		
		hipDeviceSynchronize();

	}


	//This can be made async too
	hipSetDevice(0);
	hipMemcpy(histogram, dev_histogram1, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipSetDevice(1);
	hipMemcpy(histogram2, dev_histogram2, 10 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
		printf("[%i]: %i + %i + %i = %i\n", i, histogram[i], histogram2[i], tmp[i], histogram[i] + histogram2[i] + tmp[i]);
		fprintf(output, "%i\t%i\t%i\n", histogram[i], histogram2[i], histogram[i] + histogram2[i] + tmp[i]);
	}


	//===================================================

	//Stopping the timer
	end = clock();
	exec_time = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\nExecution Time: %f\n", exec_time);

	fclose(input);
	fclose(output);

}

__global__ void computeHistogram(char* inputArr, int* histArr) {

	int globalID = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalID >= readBlockSize) {
		//printf("Returning for ID: %i", globalID);
		return;
	}

	//Return if '.', or EOL
	if (inputArr[globalID] == '\0')
		return;

	atomicAdd(&histArr[inputArr[globalID] - '0'], 1);
}

