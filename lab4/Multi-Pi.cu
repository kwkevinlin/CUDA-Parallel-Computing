
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

//#define Threads 768
#define Threads 768

#define readBlockSize 7

__global__ void computeHistogram(char*, int*);
__global__ void blankCall() {int i = 0; if (i == 0) {} };

int main(int argc, char *argv[]) {

	if (argc != 2) {
		printf("\nError: Number of arguments incorrect.\n"
			"There can only be 1 additional argument, which is the name of the file.\n"
			"Ex: ./Multi-Pi pi.txt\n"
			"Program gracefully terminated.\n");
		exit(0);
	}

	FILE *input = fopen(argv[1], "r+");
	FILE *output = fopen("freq.dat", "w+");
	if (input == NULL) {
		printf("Error: %s could not be read!\n"
			"Program gracefully terminated.\n", argv[1]);
		exit(1);
	}
	if (output == NULL) {
		printf("Error: freq.dat could not be created!\n"
				"Program gracefully terminated.\n");
		exit(1);
	}

	//------------------------------------------------------------------------------------//

	//Timer and Dummy Call
	blankCall<<<1, 1>>>();

	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipStream_t stream2;
	hipStreamCreate(&stream2); 

	clock_t start, end;
	double exec_time;
	start = clock();

	//===================================================

	char inputString[readBlockSize];
	int histogram[10] = {0}, count = 1;

	char *dev_inputString; 
	int *dev_histogram;
	hipMalloc((void**)&dev_inputString, sizeof(char) * readBlockSize);
	hipMalloc((void**)&dev_histogram, sizeof(int) * 10);
	//cudaHostAlloc((void**) &inputString, sizeof(char) * readBlockSize, cudaHostAllocDefault);
	//cudaHostAlloc((void**) &histogram, sizeof(int) * 10, cudaHostAllocDefault);

	hipSetDevice(0);
	hipMemcpy(dev_histogram, &histogram, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipSetDevice(1);
	hipMemcpy(dev_histogram, &histogram, 10 * sizeof(int), hipMemcpyHostToDevice);

	while(fgets(inputString, readBlockSize, input) != NULL) {
		printf("\t%s\n", inputString);
		hipSetDevice(0);
		hipMemcpy(dev_inputString, &inputString, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
		computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString, dev_histogram);
		// cudaMemcpyAsync(dev_inputString, &inputString, readBlockSize * sizeof(char), cudaMemcpyHostToDevice, stream1);
		// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, stream1>>>(dev_inputString, dev_histogram);

		if (fgets(inputString, readBlockSize, input) != NULL) {
			printf("Inside Nest\n");
			hipSetDevice(1);
			hipMemcpy(dev_inputString, &inputString, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
			computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString, dev_histogram);
			// cudaMemcpyAsync(dev_inputString, &inputString, readBlockSize * sizeof(char), cudaMemcpyHostToDevice, stream1);
			// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, stream1>>>(dev_inputString, dev_histogram);
		
		}

		hipDeviceSynchronize();

		printf("GPUs Synchronized (%i)\n", count);
		count++;

	}
	int histogram2[10] = {0};
	hipSetDevice(0);
	hipMemcpy(histogram, dev_histogram, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipSetDevice(1);
	hipMemcpy(histogram2, dev_histogram, 10 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
		printf("[%i]: %i\n", i, histogram[i] + histogram2[i]);
	}



	//===================================================

	//Stopping the timer
	end = clock();
	exec_time = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\nExecution Time: %f\n", exec_time);

	fclose(input);
	fclose(output);

}

__global__ void computeHistogram(char* inputArr, int* histArr) {

	int globalID = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalID >= readBlockSize)
		return;

	//Return if '.', or EOL
	if (inputArr[globalID] == '.' || inputArr[globalID] == '\0')
		return;
	printf("Reading: %c\n", inputArr[globalID]);
	atomicAdd(&histArr[inputArr[globalID] - '0'], 1);
	// printf("Current [%i] = %i\n", inputArr[globalID] - '0', histArr[inputArr[globalID] - '0']);

	// for (int i = 0; i < sizeof(inputArr)/sizeof('c') - 1; i++) {
	// 		if (inputArr[i] == '.') continue;
	// 		else if (inputArr[i] == '\0') { 
	// 			*exitFlag = 1; 
	// 			return; 
	// 		}
	// 		printf("Reading: %c, %i\n", inputArr[i], i);
	// 		histArr[inputArr[i] - '0']++;
	// 	}
}

