
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

//#define Threads 768
#define Threads 500

#define readBlockSize 100000

__global__ void computeHistogram(char*, int*);
__global__ void blankCall() {int i = 0; if (i == 0) {} };

int main(int argc, char *argv[]) {

	if (argc != 2) {
		printf("\nError: Number of arguments incorrect.\n"
			"There can only be 1 additional argument, which is the name of the file.\n"
			"Ex: ./Multi-Pi pi.txt\n"
			"Program gracefully terminated.\n");
		exit(0);
	}

	FILE *input = fopen(argv[1], "r+");
	FILE *output = fopen("freq.dat", "w+");
	if (input == NULL) {
		printf("Error: %s could not be read!\n"
			"Program gracefully terminated.\n", argv[1]);
		exit(1);
	}
	if (output == NULL) {
		printf("Error: freq.dat could not be created!\n"
				"Program gracefully terminated.\n");
		exit(1);
	}

	//------------------------------------------------------------------------------------//

	// cudaStream_t stream1;
	// cudaStreamCreate(&stream1);
	// cudaStream_t stream2;
	// cudaStreamCreate(&stream2); 

	clock_t start, end;
	double exec_time;
	start = clock();

	//===================================================

	//char inputString[readBlockSize];
	char* inputString = (char*)malloc(sizeof(char) * readBlockSize);
	int histogram[10] = {0}, histogram2[10] = {0}, count = 1; 

	char *dev_inputString1;
	char *dev_inputString2; 
	int *dev_histogram1;
	int *dev_histogram2;

	hipSetDevice(0);
		blankCall<<<1, 1>>>();
		hipMalloc((void**)&dev_inputString1, sizeof(char) * readBlockSize);
		hipMalloc((void**)&dev_histogram1, sizeof(int) * 10);
		hipMemcpy(dev_histogram1, &histogram, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipSetDevice(1);
		blankCall<<<1, 1>>>();
		hipMalloc((void**)&dev_inputString2, sizeof(char) * readBlockSize);
		hipMalloc((void**)&dev_histogram2, sizeof(int) * 10);
		hipMemcpy(dev_histogram2, &histogram, 10 * sizeof(int), hipMemcpyHostToDevice);

	//cudaHostAlloc((void**) &inputString, sizeof(char) * readBlockSize, cudaHostAllocDefault);
	//cudaHostAlloc((void**) &histogram, sizeof(int) * 10, cudaHostAllocDefault);

	while(fgets(inputString, readBlockSize, input) != NULL) {
		printf("\t%s\n", inputString);
		hipSetDevice(0);
		hipMemcpy(dev_inputString1, &inputString, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
		computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString1, dev_histogram1);
		// cudaMemcpyAsync(dev_inputString, &inputString, readBlockSize * sizeof(char), cudaMemcpyHostToDevice, stream1);
		// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, stream1>>>(dev_inputString, dev_histogram);

		if (fgets(inputString, readBlockSize, input) != NULL) {
			printf("\t%s\n", inputString);
			hipSetDevice(1);
			hipMemcpy(dev_inputString2, &inputString, readBlockSize * sizeof(char), hipMemcpyHostToDevice);
			computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads>>>(dev_inputString2, dev_histogram2);
			// cudaMemcpyAsync(dev_inputString, &inputString, readBlockSize * sizeof(char), cudaMemcpyHostToDevice, stream1);
			// computeHistogram<<<(int)ceil(readBlockSize / Threads) + 1, Threads, stream1>>>(dev_inputString, dev_histogram);
		}

		hipDeviceSynchronize();

		printf("GPUs Synchronized (%i)\n", count);
		count++;

	}

	hipSetDevice(0);
	hipMemcpy(histogram, dev_histogram1, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipSetDevice(1);
	hipMemcpy(histogram2, dev_histogram2, 10 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
		printf("[%i]: %i\n", i, histogram[i] + histogram2[i]);
	}



	//===================================================

	//Stopping the timer
	end = clock();
	exec_time = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\nExecution Time: %f\n", exec_time);

	fclose(input);
	fclose(output);

}

__global__ void computeHistogram(char* inputArr, int* histArr) {

	int globalID = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalID >= readBlockSize)
		return;

	//Return if '.', or EOL
	if (inputArr[globalID] == '.' || inputArr[globalID] == '\0')
		return;
	printf("Reading: %c\n", inputArr[globalID]);
	atomicAdd(&histArr[inputArr[globalID] - '0'], 1);

	// printf("Current [%i] = %i\n", inputArr[globalID] - '0', histArr[inputArr[globalID] - '0']);

	// for (int i = 0; i < sizeof(inputArr)/sizeof('c') - 1; i++) {
	// 		if (inputArr[i] == '.') continue;
	// 		else if (inputArr[i] == '\0') { 
	// 			*exitFlag = 1; 
	// 			return; 
	// 		}
	// 		printf("Reading: %c, %i\n", inputArr[i], i);
	// 		histArr[inputArr[i] - '0']++;
	// 	}
}

